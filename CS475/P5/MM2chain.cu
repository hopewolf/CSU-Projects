#include "hip/hip_runtime.h"
///
/// MM2chain.cu
/// By Waruna Ranasinghe 
/// Created: 18 Aug 2017
/// Last Modified: 

///NOTE: Please read all the comments before modifying the file. Failure to maintain
//the required variable names and function names will be penalized. 

//Computes the matrix multiplication of sequence of 2x2 matrices.
//The operation is associative but not commutative. Therefore you cannot
//change the order of matrix multiplications

//Includes
#include <stdio.h>
#include "MM2chainKernel.h"
#include "timer.h"

//defines

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                      \
  hipError_t _m_cudaStat = value;                    \
  if (_m_cudaStat != hipSuccess) {                   \
    fprintf(stderr, "Error: %s at line %d in file %s\n",          \
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
    exit(1);                              \
  } }

#define epsilon (float)1e-5

///
/// main
///
int main(int argc, char** argv) {
	int G; //number of thread blocks in the grid	
	int n; //number of 2x2 matrices

	//host variables
	float result[4]; // your final answer should be assigned to this variable
	float result_temp[4];
	float *h_A; //input 2x2 matrix sequence. Stored in a 1D array
	float *h_matmults; //partial output matmults per thread block
	//device variables
	float *d_A; //input 2x2 matrix sequence. Stored in a 1D array
	float *d_matmults; //partial output matmults per thread block
	
	if (argc < 2) {
		printf("Usage: %s G\n\tG - number of thread blocks in the grid\n", argv[0]);
		exit(0);
	} 

	//init size params
	G = atoi(argv[1]);
	n=B*G*C;
    // UNCOMMENT THIS WHEN CHECKIN
// 	printf("Number of threads blocks: %d\nNumber of threads per block: %d\nNumber of matrices per thread: %d\n", G, B, C);
	//allocating memory for host variables
	h_A = (float *)malloc(n*4*sizeof(float));
	if (h_A==NULL) {
		fprintf(stderr, "Failed to allocate host array h_A!\n"); 
		exit(-1);
	}

	h_matmults = (float *)malloc(G*4*sizeof(float));
	if (h_matmults==NULL) {
		fprintf(stderr, "Failed to allocate host array h_matmults!\n");
		exit(-1);
	}

	//initializing input matrices
	//  We repeat the following sequence of matrices      
	//  |1.0 0.0| |4.0 3.0| |1.0  0.0| |0.4 -0.3|
	//  |0.5 1.0| |0.0 2.5| |-0.5 1.0| |0.0 0.25|
	//         
	for (int i=0; i<n*4; i+=16) {
		h_A[i]=1.0f;
		h_A[i+1]=0.0f;
		h_A[i+2]=0.5f;
		h_A[i+3]=1.0f;
		h_A[i+4]=4.0f;
		h_A[i+5]=3.0f;
		h_A[i+6]=0.0f;
		h_A[i+7]=2.5f;
		h_A[i+8]=1.0f;
		h_A[i+9]=0.0f;
		h_A[i+10]=-0.5f;
		h_A[i+11]=1.0f;
		h_A[i+12]=0.4f;
		h_A[i+13]=-0.3f;
		h_A[i+14]=0.0f;
		h_A[i+15]=0.25f;
	}	

	CUDA_CHECK_RETURN(hipSetDevice(0));


	//Allocating the device memory for input vector. 
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_A, n*4*sizeof(float)));	

	//allocate device memory for the partial output
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_matmults, G*4*sizeof(float)));	

	initialize_timer ();
	start_timer();
	//copy the host input matrices array h_A to the device input array d_A in device
	//memory	
	CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, n*4*sizeof(float), hipMemcpyHostToDevice));

	float time_input, time_gpu, time_out; 
  /* Start Timer */
	//initialize_timer ();
	//start_timer();

	stop_timer();
	time_input=elapsed_time ();
	reset_timer();
	start_timer();
	//Launch the reduce CUDA kernel with G blocks per grid and B threads per
	//block
	multiply<<<G, B>>>(d_A, d_matmults);

	//check whether there were errors while launching the CUDA kernel
	CUDA_CHECK_RETURN(hipGetLastError());

	//wait for the kernel to finish
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	/* stop timer */
	stop_timer();
	time_gpu=elapsed_time ();
	reset_timer();
	start_timer();

	//Copy partial results back to host
	CUDA_CHECK_RETURN(hipMemcpy(h_matmults, d_matmults, G*4*sizeof(float), hipMemcpyDeviceToHost));

	result[0] = 1.0f;
	result[1] = 0.0f;
	result[2] = 0.0f;
	result[3] = 1.0f;
	//compute final answer by multiplying the partial answers of threads from
	//GPU
	for (int i=0; i<G; i++) {
		result_temp[0] = result[0]*h_matmults[i*4] + result[1]*h_matmults[i*4+2];
		result_temp[1] = result[0]*h_matmults[i*4+1] + result[1]*h_matmults[i*4+3];
		result_temp[2] = result[2]*h_matmults[i*4] + result[3]*h_matmults[i*4+2];
		result_temp[3] = result[2]*h_matmults[i*4+1] + result[3]*h_matmults[i*4+3];
		result[0] = result_temp[0];
		result[1] = result_temp[1];
		result[2] = result_temp[2];
		result[3] = result_temp[3];
	}

	stop_timer();
	time_out=elapsed_time ();
    //THIS IS THE PRINT THAT SHOULD REMAIN WHEN CHECKIN!!!
	//printf("Output:\n%f %f\n%f %f\nTime to copy input data: %f s\nTime to compute[GPU]: %f s\nTime to compute output: %f s\n", result[0], result[1], result[2], result[3], time_input, time_gpu, time_out);
    //THIS IS THE PRINT FOR GENERATING report
    printf("%f \n", time_gpu);
//	printf("Result: %f , Elapsed time [GPU]: %f , all: %f\n", result, time);

//	if (result[0]-1.0f < epsilon && result[1]-0.0f < 0.2f && result[2]-0.0f < epsilon && result[3]-1.0f < epsilon) {
	if (fabsf(result[0]-1.0f) < epsilon && fabsf(result[1]-0.0f) < 0.2f && fabsf(result[2]-0.0f) < epsilon && fabsf(result[3]-1.0f) < epsilon) {
        // UNCOMMENT BEFORE TURNING IN!
// 		printf("Test passed.\n");
	} else {
		printf("Test failed.\n");
	}


	//cleaning up
	free(h_A);
	free(h_matmults);

	return 0;
}

