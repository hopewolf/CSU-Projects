
#include <hip/hip_runtime.h>
#include <stdio.h> 

///
/// MM2chainkernel01.cu
/// By Luke Burford 
/// Created: 1 Nov 2018

#define B 32 // threads per block 
#define C 60 // matracies proccessed per block 

__global__ void multiply(const float* A, float* tempoMult){ 
    int i; 
    
    
    float MxM00 = 0; 
    float MxM01 = 0; 
    float MxM10 = 0; 
    float MxM11 = 0; 
    
    float TH00 = 0; 
    float TH01 = 0; 
    float TH10 = 0; 
    float TH11 = 0; 
    
    __shared__ float SharedAMat[(B * C * 4)]; 

    #pragma unroll 
    for( i = 0 ; i < 4 * C ; ++i){ 
        SharedAMat[i*B + threadIdx.x] = A[(blockIdx.x * B *  4 * C )+  i*B + threadIdx.x]; 
    } 
    
    __syncthreads(); 
    
    
    TH00 = SharedAMat[threadIdx.x * 4 * C  + 0]; 
    TH01 = SharedAMat[threadIdx.x * 4 * C  + 1]; 
    TH10 = SharedAMat[threadIdx.x * 4 * C  + 2]; 
    TH11 = SharedAMat[threadIdx.x * 4 * C  + 3]; 
    
    #pragma unroll 
    for (i = 4 + (threadIdx.x * 4 * C) ; i < C * 4 +  (threadIdx.x * 4 * C); i+=4){ 
        MxM00 = (TH00 * SharedAMat[i])   + (TH01 * SharedAMat[i+2]); 
        MxM01 = (TH00 * SharedAMat[i+1]) + (TH01 * SharedAMat[i+3]); 
        MxM10 = (TH10 * SharedAMat[i])   + (TH11 * SharedAMat[i+2]); 
        MxM11 = (TH10 * SharedAMat[i+1]) + (TH11 * SharedAMat[i+3]); 
        TH00 = MxM00; 
        TH01 = MxM01; 
        TH10 = MxM10; 
        TH11 = MxM11; 
    } 
    
    SharedAMat[threadIdx.x * 4 * C  + 0] = TH00; 
    SharedAMat[threadIdx.x * 4 * C  + 1] = TH01; 
    SharedAMat[threadIdx.x * 4 * C  + 2] = TH10; 
    SharedAMat[threadIdx.x * 4 * C  + 3] = TH11; 

    __syncthreads(); 
    
    if (threadIdx.x == 0){ 
        #pragma unroll 
        for (i = C *  4; i < B * 4 * C; i+=C * 4){ 
            MxM00 = (TH00 * SharedAMat[i])   + (TH01 * SharedAMat[i+2]); 
            MxM01 = (TH00 * SharedAMat[i+1]) + (TH01 * SharedAMat[i+3]); 
            MxM10 = (TH10 * SharedAMat[i])   + (TH11 * SharedAMat[i+2]); 
            MxM11 = (TH10 * SharedAMat[i+1]) + (TH11 * SharedAMat[i+3]); 
            TH00 = MxM00; 
            TH01 = MxM01; 
            TH10 = MxM10; 
            TH11 = MxM11; 
        } 
        tempoMult[0 + blockIdx.x * 4] = TH00; 
        tempoMult[1 + blockIdx.x * 4] = TH01; 
        tempoMult[2 + blockIdx.x * 4] = TH10; 
        tempoMult[3 + blockIdx.x * 4] = TH11; 
    }   
}
