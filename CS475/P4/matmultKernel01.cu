#include "hip/hip_runtime.h"
///
/// matmultKernel00.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman and Luke Burford
/// Assigned to: Luke Burford
/// Created: 2011-01-27
/// Last Modified: 2018-10-23 DVN
///
/// Multiplies two matrices using CUDA: A x B = C
///
/// Copy this file and modify the MatMultKernel device function for
/// each of your experiments. 
///

#include "matmultKernel.h"

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
    
    // matrix blocks
    float *Asub, *Bsub, *Csub;
    // Putting these into registers speeds access.
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;
    // these
    int block_row = blockIdx.y;
    int block_col = blockIdx.x;
    
    int ind[4];
    ind[0] = thread_row*A.stride + thread_col;
    ind[1] = thread_row*A.stride + thread_col + BLOCK_SIZE;
    ind[2] = (thread_row+BLOCK_SIZE)*A.stride + thread_col;
    ind[3] = (thread_row+BLOCK_SIZE)*A.stride + thread_col + BLOCK_SIZE;
    
    // Each THREAD BLOCK computes one sub matrix Csub of C
    // EACH THREAD creates its own matrix descriptor Csub
    Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * block_col];
    
    // Each thread computes one element of Csub in its copy of CValue
    float Cvalue1 = 0;
    float Cvalue2 = 0;
    float Cvalue3 = 0;
    float Cvalue4 = 0;
    
    // Loop over all sub matrices in block_row of A and block_col of B
    // required to compute Csub. Block multiply each pair of sub matrices
    // and accumulate results
    #pragma unroll
    for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){
        // Get Asub and Bsub descriptors
        Asub = &A.elements[A.stride * FOOTPRINT_SIZE * block_row + FOOTPRINT_SIZE * m];
        Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * block_col];
        
        // Copy ELEMENTS OF  ASub and Bsub into shared memory
        // EACH THREAD loads ONE ELEMENT of ASub and ONE of Bsub
        // Notice: it does not need to be the element it requires to
        //         compute its Cvalue, as long as all elements are 
        //         collaboratively read. 
        
        // Notice: every thread declares shared_A and shared_B in shared memory
        //         even though a thread block has only one shared_A and one shared_B
        
        
        __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
        __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
        
        // Each thread copies just one element of shared_A and one element of shared_B
        
        shared_A[thread_row][thread_col] = Asub[ind[0]];
        shared_A[thread_row][thread_col+BLOCK_SIZE] = Asub[ind[1]];
        shared_A[thread_row+BLOCK_SIZE][thread_col] = Asub[ind[2]];
        shared_A[thread_row+BLOCK_SIZE][thread_col+BLOCK_SIZE] = Asub[ind[3]];
        
        shared_B[thread_row][thread_col] = Bsub[ind[0]];
        shared_B[thread_row][thread_col+BLOCK_SIZE] = Bsub[ind[1]];
        shared_B[thread_row+BLOCK_SIZE][thread_col] = Bsub[ind[2]];
        shared_B[thread_row+BLOCK_SIZE][thread_col+BLOCK_SIZE] = Bsub[ind[3]];
        
        // Synchronize to ensure all elements are read
        __syncthreads();
        
        // Do an inproduct of one row of shared_A and one col of shared_B
        // computing one Cvalue by accumulation
        #pragma unroll
        for(int e=0; e<FOOTPRINT_SIZE; ++e) {
            Cvalue1 += shared_A[thread_row][e] * shared_B[e][thread_col];
            Cvalue2 += shared_A[thread_row][e] * shared_B[e][thread_col + BLOCK_SIZE];
            Cvalue3 += shared_A[thread_row + BLOCK_SIZE][e] * shared_B[e][thread_col];
            Cvalue4 += shared_A[thread_row + BLOCK_SIZE][e] * shared_B[e][thread_col + BLOCK_SIZE];
        }
        
        // Synchronize to ensure all Cvalues have been incremented
        // before reading in the next shared_A AND shared_B BLOCKS
        __syncthreads();
    }
    
    // Write Csub to GLOBAL memory.
    // Each thread writes its own cell value.
    Csub[ind[0]] = Cvalue1;
    Csub[ind[1]] = Cvalue2;
    Csub[ind[2]] = Cvalue3;
    Csub[ind[3]] = Cvalue4;
}

