#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <stdlib.h>

#include <sys/time.h>     //measuring performance data


void matrix_math(int N){
   
  double** mat1;
  double** mat2;
  double** ret;
  hipMallocManaged(&mat1, N*N * sizeof(float));		//cuda allocation of unified Memory  
  hipMallocManaged(&mat2, N*N * sizeof(float));
  hipMallocManaged(&ret, N * sizeof(float)); 
  
  //setting matricies to their upper bound 
  int i, j;
  for (i=0; i<N; i++)    {
    for (j=0; j<N; j++)    {
      if (i>j)   {
        mat1[i][j] = 0;
        mat2[i][j] = 0;
      }
      mat1[i][j] = mat1[i][j];
      mat2[i][j] = mat2[i][j];
    }
  }
 
 //multiplying matricies 
 int k;
 for (i = 0; i < N; i++)
 {
    for (j = i; j < N; j++)
    {
        res[i][j] = 0.0;
        for (k = 0; k < N; k++)
            res[i][j] += mat1[i][k]*mat2[k][j];
        }
    }
}


int main(int argc, char** argv)
{
    int N = 3;  // THIS IS WHERE YOU DEFINE THE SIZE OF THE MATRIX (it can only be square right now)
    int i, j;
  
    //performance measurment 
//    struct timeval t_start;
//    struct timeval t_end;
//    double etime;
  
    matrix_math(N);      
    
     // print matrix 1
    printf("mat1 matrix is \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
           printf("%d ", mat1[i][j]);
        printf("\n");
    }
    // print matrix 2
    printf("mat2 matrix is \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
           printf("%d ", mat2[i][j]);
        printf("\n");
    }
    // print result matrix
    printf("Result matrix is \n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
           printf("%d ", res[i][j]);
        printf("\n");
    }
    
    return 0;
}
