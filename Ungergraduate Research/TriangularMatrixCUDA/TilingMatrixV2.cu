#include "hip/hip_runtime.h"
/*
 *  file name: TilingMatrixV2.c
 *  NOTE: 
 *       squareMatrixMult is much more efficent than the regular multiplier
 *       currently compiling with: nvcc TilingMatrix.cu -o tileTest
 *       Device Standards for: GeForce GTX 1060 6GB
 *          total global mem size: 6078 MBytes (6373572608 bytes)
 *          total shared mem per block: 49.152 KBytes (49152 bytes)
 *       nvcc TilingMatrixV2.c -lcublas -o TilingMatrixTest
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <sys/time.h>     //measuring performance data

 #define BLOCK_SIZE 32

/**********************************************************************
function name: matrixTriUpper
description: sets a matrix to an upper bound triangle matrix
parameters: 
            &a GPU device pointer to a m X n matrix (A)
Note:
    
return: none
**********************************************************************/
__global__ void matrixTriUpper(float *a, int m, int n) {
    //setting matricies to their upper bound 
    for(int i = 0; i < m; ++i) {
        for(int j = 0; j < n; ++j) {
            if(i>j)
                a[i*n + j] = 0;
            a[i*n + j] = a[i*n + j];
        }
    }
}

/**********************************************************************
function name: cublasMatrixMult
description: dot product of two matricies using cublas function: hipblasSgemm
parameters: 
            &a GPU device pointer to a m X n matrix (A)
            &b GPU device pointer to a n X k matrix (B)
            &c GPU device output purpose pointer to a m X k matrix (C) 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
**********************************************************************/
__global__ void cublasMatrixMult(float *d_a, float *d_b, float *d_c, int m, int n, int k, hipblasStatus_t stat) {
    //hipGetErrorString((hipError_t) cudaStat);
//     if(cudaStat != hipSuccess)
//         printf("Cuda Error: %s\n", hipGetErrorString(cudaStat));
    

//     if(stat != HIPBLAS_STATUS_SUCCESS)
//         printf("Cublas Error: %s\n", cublasGetErrorString(stat));
    
    float al =1.0f; // al=1
    float bet =1.0f; // bet=1
    // matrix-matrix multiplication: d_c=al*d_a*d_b+bet*d_c
    // d_a -mxk matrix, d_b -kxn matrix, d_c -mxn matrix;
    // al, bet -scalars
    stat = hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,d_a,m,d_b,k,&bet,d_c,m);
//     if(stat != HIPBLAS_STATUS_SUCCESS)
//         printf("Cublas Error: %s\n", cublasGetErrorString(stat));
    
    hipblasDestroy(handle); // destroy CUBLAS context
}

/**********************************************************************
function name: matrixCpy
description: Breaks up 2 matracies into smaller chunks, and Device to be multiplied
parameters: 
            &a GPU device pointer to a m X n matrix (A)
            &b GPU device pointer to a n X k matrix (B)
            &c GPU device output purpose pointer to a m X k matrix (C) 
            to store the result
return: none
**********************************************************************/
void matrixCpy(float *a, float *b, float *c, int m, int n, int k) {

}

/**********************************************************************
function name: cublasGetErrorString
description: gets the cublas string error codes for printing
parameters: 
            error a cublas error status enum
return: char pointer (string)
TODO: Fix the return type
**********************************************************************/
const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}


/**********************************************************************
function name: main
description: test and compare
parameters: 
            none
return: none
**********************************************************************/
int main(int argc, char** argv) {
    hipError_t cudaStat; // hipMalloc status
    int m=4;// a - mxk matrix
    int n=4;// b - kxn matrix
    int k=4;// c - mxn matrix
    // Set status variables
    
    // Allocate memory in host RAM
    float *a; // mxk matrix a on the host
    float *b; // kxn matrix b on the host
    float *c; // mxn matrix c on the host
    a = (float*) malloc(m*k* sizeof(float)); // host memory for a
    b = (float*) malloc(k*n* sizeof(float)); // host memory for b
    c = (float*) malloc(m*n* sizeof(float)); // host memory for c
    
    /* Assign Random Variables to the matrecies */
    srand(3333);
    // random initialize matrix A [mxn]
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] =((float)rand()/(float)(RAND_MAX))*1024;
        }
    }

    // random initialize matrix B [kxn]
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            b[i * k + j] = ((float)rand()/(float)(RAND_MAX))*1024;
        }
    }
    
    // matracies on the device
    float *d_a; // d_a - a on the device
    float *d_b; // d_b - b on the device
    float *d_c; // d_c - c on the device
    cudaStat = hipMalloc((void**)&d_a,m*k*sizeof(*a)); // device memory alloc for a
    cudaStat = hipMalloc((void**)&d_b,k*n*sizeof(*b)); // device memory alloc for b
    cudaStat = hipMalloc((void**)&d_c,m*n*sizeof(*c)); // device memory alloc for c
    
    stat = hipblasCreate(&handle); // initialize CUBLAS context
    // copy matrices from the host to the device
    stat = hipblasSetMatrix(m,k,sizeof(*a),a,m,d_a,m); //a -> d_a
    stat = hipblasSetMatrix(k,n,sizeof(*b),b,k,d_b,k); //b -> d_b
    stat = hipblasSetMatrix(m,n,sizeof(*c),c,m,d_c,m); //c -> d_c
    
    // on host set the two matracies to triangles
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    printf("Calculating...\n\n");
    // Launch kernel
    matrixTriUpper<<<dimGrid, dimBlock>>>(d_a, m, n);
    matrixTriUpper<<<dimGrid, dimBlock>>>(d_b, n, k);
    cublasMatrixMult<<<dimGrid, dimBlock>>>(d_a,d_b,d_c,m,n,k);
    
    stat = hipblasGetMatrix(m,n,sizeof(*c),d_c,m,c,m,stat); // cp d_c -> c
    int i,j;
    // print matrix A
    printf("matA matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            //printf("[%d][%d]:%d, ", i, j, a[i*k + j]);
            printf(" %f ", a[i*k + j]);
        }
        printf("\n");
    }
    // print matrix B
    printf("\nmatB matrix: \n");
    for (i = 0; i < n; i++) {
        for (j = 0; j < k; j++) {
            //printf("[%d][%d]:%d, ", i, j, b[i*k + j]);
            printf(" %f ", b[i*k + j]);
        }
        printf("\n");
    }
    // print result matrix
    printf("\nResult matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < k; j++) {
                //printf("[%d][%d]:%d, ", i, j, c[i*k + j]);
                printf(" %f ", c[i*k + j]);
            }
        printf("\n");
    }
    
    // free memory
    hipFree(a);
    hipFree(a);
    hipFree(a);
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
