/*
 *  file name: BlockingMat.cu
 *  NOTE: 
 *       squareMatrixMult is much more efficent than the regular multiplier
 *       currently compiling with: nvcc TilingMatrix.cu -o tileTest
 *       Device Standards for: GeForce GTX 1060 6GB
 *          total global mem size: 6078 MBytes (6373572608 bytes)
 *          total shared mem per block: 49.152 KBytes (49152 bytes)
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include <stdlib.h>

#include <sys/time.h>     //measuring performance data

#define BLOCK_SIZE 32


/**********************************************************************
function name: main
description: test and compare
parameters: 
            none
return: none
**********************************************************************/
int main(int argc, char** argv) {
    int m=4;
    int n=4;
    int k=4;

    // Allocate memory in host RAM
    float *copyA, *copyB, *copyC;
    hipHostMalloc((void**) &copyA, sizeof(float)*m*n, hipHostMallocDefault); // copied matrix is m x n
    hipHostMalloc((void**) &copyB, sizeof(float)*n*k, hipHostMallocDefault); // copied matrix is n x k
    hipHostMalloc((void**) &copyC, sizeof(float)*m*k, hipHostMallocDefault); // copied matrix is m x k
    
    // float x = (float)rand()/(float)(RAND_MAX/a);
    // random initialize matrix A
    float val = 1;
    for (int i = 0; i < m; ++i) {
        for (int j = i; j < n; ++j) {
//             copyA[i * n + j] =((float)rand()/(float)(RAND_MAX)) * 1024;
            copyA[i * n + j] = val;
            ++val;
        }
    }
    val = 1;
    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = i; j < k; ++j) {
            copyB[i * k + j] = val;
            ++val;
        }
    }
    
    // Allocate memory space on the device 
    float *matA, *matB, *matC;
    hipMalloc((void**) &matA, sizeof(float)*m*n); // matrix is m x n
    hipMalloc((void**) &matB, sizeof(float)*n*k); // matrix is n x k
    hipMalloc((void**) &matC, sizeof(float)*m*k); // matrix is m x k
    
    // copy matrix A and B from host to device memory
    hipMemcpy(matA, copyA, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(matB, copyB, sizeof(float)*n*k, hipMemcpyHostToDevice);
    
    printf("size of matA %dX%d: %zu bytes\n", m,n,(sizeof(float)*m*n));
    printf("size of matB %dX%d: %zu bytes\n", n,k,(sizeof(float)*n*k));
    printf("size of matC %dX%d: %zu bytes\n", m,k,(sizeof(float)*m*k));
    printf("total bytes allocated to mem: %zu bytes ", ((sizeof(float)*m*n)+(sizeof(float)*n*k)+ (sizeof(float)*m*k)));
    printf("(~%zu MBytes)\n\n", (((sizeof(float)*m*n)+(sizeof(float)*n*k)+(sizeof(float)*m*k)) / 1000000)); // get megabytes of the allocated arrays

    
    //prints the matricies
    // printf("[%d][%d]:%d, ", i, j, copyC[i*k + j]); //Another possible way to print the matrix
    //if the debug flag is on it will print the first two product arrays as well
    int i,j;
    // print matrix A
    printf("matA matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < n; j++) {
            //printf("[%d][%d]:%d, ", i, j, copyA[i*k + j]);
            printf(" %f ", copyA[i*k + j]);
        }
        printf("\n");
    }
    // print matrix B
    printf("\nmatB matrix: \n");
    for (i = 0; i < n; i++) {
        for (j = 0; j < k; j++) {
            //printf("[%d][%d]:%d, ", i, j, copyB[i*k + j]);
            printf(" %f ", copyB[i*k + j]);
        }
        printf("\n");
    }
    
    // print result matrix
    printf("\nResult matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < k; j++) {
            //printf("[%d][%d]:%d, ", i, j, copyC[i*k + j]);
            printf(" %f ", copyC[i*k + j]);
        }
        printf("\n");
    }
    
    // free memory
    hipFree(matA);
    hipFree(matB);
    hipFree(matC);
    hipHostFree(copyA);
    hipHostFree(copyB);
    hipHostFree(copyC);
    return 0;
}
