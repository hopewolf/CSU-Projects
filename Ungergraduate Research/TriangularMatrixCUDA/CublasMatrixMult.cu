// nvcc 036 sgemm .c -lcublas
# include <stdio.h>
# include <stdlib.h>
# include <hip/hip_runtime.h>
# include "hipblas.h"
#include <sys/time.h>     //measuring performance data
# define IDX2C(i ,j , ld ) ((( j )*( ld ))+( i ))
# define m 32000 // a - mxk matrix
# define n 32000 // b - kxn matrix
# define k 32000 // c - mxn matrix

/**********************************************************************
 * function name: cublasGetErrorString
 * description: gets the cublas string error codes for printing
 * parameters: 
 * error a cublas error status enum
 * return: char pointer (string)
 **********************************************************************/
const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "CUBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "CUBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "CUBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "CUBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

int main ( void ){
    struct timeval time;
    double elapsed_time;
    hipError_t cudaStat ; // cudaMalloc status
    hipblasStatus_t stat ; // CUBLAS functions statusx
    hipblasHandle_t handle ; // CUBLAS context
    int i,j; // i-row index ,j- column index
    float *a; // mxk matrix a on the host
    float *b; // kxn matrix b on the host
    float *c; // mxn matrix c on the host
    gettimeofday(&time, NULL);
    elapsed_time = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000);
    a=(float*)malloc(m*k*sizeof(float)); // host memory for a
    b=(float*)malloc(k*n*sizeof(float)); // host memory for b
    c=(float*)malloc(m*n*sizeof(float)); // host memory for c
    // define an mxk matrix a column by column
    int ind =11; // a:
    // print a row by row
//     printf ("a:\n");
    int val = 1;
    // random initialize matrix A [mxk]
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] =val++;
//             printf("%5.0f", a[i * n + j]);
        }
//         printf("\n");
    }
    // define a kxn matrix b column by column
    

    // print b row by row
    val = 1;
//     printf ("b:\n");
    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < n; ++j) {
            b[i * n + j] =val++;
//             printf("%5.0f", b[i * n + j]);
        }
//         printf("\n");
    }
    // define an mxn matrix c column by column
    // print c row by row
//     printf ("c:\n");
    val = 1;
//     for (int i = 0; i < m; ++i) {
//         for (int j = 0; j < n; ++j) {
//             c[i * n + j] =val++;
//             printf("%5.0f", c[i * n + j]);
//         }
//         printf("\n");
//     }
    // on the device
    float *d_a; // d_a - a on the device
    float *d_b; // d_b - b on the device
    float *d_c; // d_c - c on the device
    cudaStat = hipMalloc((void**)&d_a,m*k*sizeof(*a)); // device memory alloc for a
    cudaStat = hipMalloc((void**)&d_b,k*n*sizeof(*b)); // device memory alloc for b
    cudaStat = hipMalloc((void**)&d_c,m*n*sizeof(*c)); // device memory alloc for c
    stat = hipblasCreate(&handle); // initialize CUBLAS context
//     printf("Cublas Create Error: %s\n", cublasGetErrorString(stat));
    // copy matrices from the host to the device
    //cudaGetErrorString((cudaError_t) cudaStat);
    stat = hipblasSetMatrix(m,k,sizeof(*a),a,m,d_a,m); //a -> d_a
    stat = hipblasSetMatrix(k,n,sizeof(*b),b,k,d_b,k); //b -> d_b
    stat = hipblasSetMatrix(m,n,sizeof(*c),c,m,d_c,m); //c -> d_c
    float al =1.0f; // al =1
    float bet =0.0f; // bet =0
    // matrix - matrix multiplication : d_c = al*d_a *d_b + bet *d_c
    // d_a -mxk matrix , d_b -kxn matrix , d_c -mxn matrix ;
    // al ,bet -scalars
    stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,d_a,m,d_b,k,&bet,d_c,m);
    stat = hipblasGetMatrix (m,n, sizeof (*c) ,d_c ,m,c,m); // cp d_c - >c
//     printf ("c after Sgemm :\n");
//     for (i = 0; i < m; i++) {
//         for (j = 0; j < n; j++) {
//             //printf("[%d][%d]:%d, ", i, j, c[i*k + j]);
//             printf(" %7.0f ", c[i*n + j]);
//         }
//         printf("\n");
//     }
    hipFree(d_a); // free device memory
    hipFree(d_b); // free device memory
    hipFree(d_c); // free device memory
    hipblasDestroy(handle); // destroy CUBLAS context
    gettimeofday(&time, NULL);
    elapsed_time = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000) - elapsed_time;
    printf("Execution time : %lf sec.\n", elapsed_time);
    free(a); // free host memory
    free(b); // free host memory
    free(c); // free host memory
    return EXIT_SUCCESS;
}
// a:
// 11 17 23 29 35
// 12 18 24 30 36
// 13 19 25 31 37
// 14 20 26 32 38
// 15 21 27 33 39
// 16 22 28 34 40
// b:
// 11 16 21 26
// 12 17 22 27
// 13 18 23 28
// 14 19 24 29
// 15 20 25 30
// c:
// 11 17 23 29
// 12 18 24 30
// 13 19 25 31
// 14 20 26 32
// 15 21 27 33
// 16 22 28 34
// c after Sgemm :
// 1566 2147 2728 3309
// 1632 2238 2844 3450
// 1698 2329 2960 3591 // c=al*a*b+bet *c
// 1764 2420 3076 3732
// 1830 2511 3192 3873
// 1896 2602 3308 4014
