/*
 *  file name: TilingMatrixV2.c
 *  NOTE: 
 *       squareMatrixMult is much more efficent than the regular multiplier
 *       currently compiling with: nvcc TilingMatrix.cu -o tileTest
 *       Device Standards for: GeForce GTX 1060 6GB
 *          total global mem size: 6078 MBytes (6373572608 bytes)
 *          total shared mem per block: 49.152 KBytes (49152 bytes)
 *       nvcc TilingMatrixV2.c -lcublas -o TilingMatrixTest
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <sys/time.h>     //measuring performance data

#define BLOCK_SIZE 32

/**********************************************************************
 * function name: matrixTriUpper
 * description: sets a matrix to an upper bound triangle matrix
 * parameters: 
 * &a GPU device pointer to a m X n matrix (A)
 * Note:
 * 
 * return: none
 **********************************************************************/
__global__ void matrixTriUpper(float *a, int m, int n) {
    //setting matricies to their upper bound 
    for(int i = 0; i < m; ++i) {
        for(int j = 0; j < n; ++j) {
            if(i>j)
                a[i*n + j] = 0;
            a[i*n + j] = a[i*n + j];
        }
    }
}

/**********************************************************************
 * function name: cublasGetErrorString
 * description: gets the cublas string error codes for printing
 * parameters: 
 * error a cublas error status enum
 * return: char pointer (string)
 **********************************************************************/
const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "CUBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "CUBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "CUBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "CUBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "CUBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "CUBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "CUBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "CUBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

/**************************************************************
 * function name: storeC
 * description: copies the final answers of tileC back to the corresponding indices of of Matrix C
 * 
 * parameters:
 *            &tilec          pointer to pre-allocated (tileLength X tileLength) matrix
 *            &matrixc        pointer to large (m X m)matrix B 
 *            int tileLength  predefined length of tile
 *            int i           caller outer loop value (helps define starting ROW index for tile)
 *            int j           caller inner loop value (helps define starting COLUMN for tile)
 * 
 ****************************************************************/
void storeC (float *tileC, float *matrixC, int tileLength, int i, int j, int numTiles){
    //pointer declarations
    for(int Ti = (tileLength*i); Ti < (tileLength*i)+tileLength; Ti++){
        for(int Tj = (tileLength*j); Tj < (tileLength*j) + tileLength; Tj++ ){
            matrixC[(Ti * numTiles) + Tj] = tileC[(Ti *numTiles *tileLength)+Tj];  
            //             printf("[%0.1f] ", tileC[(Ti *numTiles *tileLength)+Tj]);
        }
        //         printf("\n");
    }
    //     printf("\n");
    
    
}

/**************************************************************
 * function name: fillA
 * description: populates TileA with elements of matrix A that correspond to the to the correct starting indices and boundries of the tile.
 * 
 * parameters:
 *           &tileA          pointer to pre-allocated tileLength X tileLength matrix
 *           &matrixA        pointer to large matrix A 
 *           int tileLength  predefined length of tile
 *           int i           caller outer loop value (helps define starting ROW index for tile)
 *           int j           caller inner loop value (helps define starting COLUMN for tile)
 * 
 ****************************************************************/
void fillA(float *tileA, float *matrixA, int tileLength, int i, int j, int numTiles){
    //pointer declarations
    
    for(int Ti = (tileLength*i); Ti < (tileLength*i)+tileLength; Ti++){
        for(int Tj = (tileLength*j); Tj < (tileLength*j) + tileLength; Tj++ ){
            tileA[(Ti * tileLength) + Tj] = matrixA[(Ti *numTiles *tileLength) + Tj]; 
            //             printf("[%0.1f] ", tileA[(Ti * tileLength) + Tj]);
        }
        //         printf("\n");
    }
    //     printf("\n");
    
}

/**************************************************************
 * function name: fillB
 * description: populates TileB with elements of matrix B that correspond to the to the correct starting indices and boundries of the
 * tile.
 * 
 * parameters:
 *           &tileB          pointer to pre-allocated (tileLength X tileLength) matrix
 *           &matrixB        pointer to large (m X m)matrix B 
 *           int tileLength  predefined length of tile
 *           int i           caller outer loop value (helps define starting COLUMN index for tile)
 *           int j           caller inner loop value (helps define starting ROW for tile)
 * 
 ****************************************************************/
void fillB(float *tileB, float *matrixB, int tileLength, int i, int j, int numTiles){
    //pointer declarations
    
    for(int Ti = (tileLength*j); Ti < (tileLength*j)+tileLength; Ti++){
        for(int Tj = (tileLength*i); Tj < (tileLength*i) + tileLength; Tj++ ){
            
            
            tileB[Ti * tileLength + Tj] = matrixB[Ti * numTiles* tileLength + Tj]; 
            //             printf("[%0.1f] ", tileB[Ti * tileLength + Tj]);
        }
        //         printf("\n");
    }
    //     printf("\n");
}

/**********************************************************************
 * function name: matrixCpy
 * description: Iterates through large (m X m) matricies A and B, continually creating smaller (tileLength * tileLength) matricies Ta and Tb that will be used by device to produce matrix C containing computed answers of MM of matrices A and B.  
 * parameters: handle
 * &a              GPU device pointer to a m X m matrix (A)
 * &b              GPU device pointer to a m X m matrix (B)
 * &c              GPU device output purpose pointer to a m X m matrix (C) 
 * int tileLength  predefined max length of tile
 * int m           # of tiles that divide the length of matrices A & B
 * 
 * return: none
 * TODO implement kernel calls of cuBlas and TMM, implement another function or code that tranfers results of C tile to matrix C. FIGURE OUT WHY fillA and fillB piss off the compiler
 **********************************************************************/
void matrixCpy(float *a, float *b, float *c, int tileLength, int m) {
    hipError_t cudaStat; // cudaMalloc & cudaFree status
    hipblasStatus_t stat; // CUBLAS functions statusx
    hipblasHandle_t handle; // CUBLAS context
    float al =1.0f; // al =1
    float bet =1.0f; // bet =1
    float *Ta,*Tb,*Tc, *d_a, *d_b, *d_c; // device and host TILE memory declaration
    struct timeval copyTime;
    double copy_elapsed_time;
    struct timeval mathTime;
    double math_elapsed_time;
    int avgCopTemp = 0;
    int avgMathTemp = 0;
    int numberofIter = 0;
    
    stat = hipblasCreate(&handle); // initialize CUBLAS context
    if(stat != HIPBLAS_STATUS_SUCCESS)
        printf("Cublas Create Error: %s\n", cublasGetErrorString(stat));
    //Host memory alocation
    Ta = (float*) malloc(tileLength*tileLength*sizeof(float)*3); // host tile memory alloc for a
    Tb = (float*) malloc(tileLength*tileLength*sizeof(float)*3); // host tile memory alloc for b
    Tc = (float*) malloc(tileLength*tileLength*sizeof(float)*3); // host tile memory for c
    
    //Device memory allocations
    cudaStat = hipMalloc((void**)&d_a,tileLength*tileLength*sizeof(*a)); // device memory alloc for a
    if(cudaStat != hipSuccess)
        printf("Cuda A Malloc: %s\n", hipGetErrorString(cudaStat));
    cudaStat = hipMalloc((void**)&d_b,tileLength*tileLength*sizeof(*b)); // device memory alloc for b
    if(cudaStat != hipSuccess)
        printf("Cuda B Malloc: %s\n", hipGetErrorString(cudaStat));
    cudaStat = hipMalloc((void**)&d_c,tileLength*tileLength*sizeof(*c)); // device memory alloc for c
    if(cudaStat != hipSuccess)
        printf("Cuda malloc Error: %s\n", hipGetErrorString(cudaStat));
    
    for(int i = 0; i < m; i++)
    {
        //memcpy of tile C for host to device (POSSIBLE AREA FOR TIMING)
        //         cudaStat = cudaMemcpy(d_c, Tc, tileLength*tileLength*sizeof(float), cudaMemcpyHostToDevice);
        //         if(cudaStat != cudaSuccess)
        //             printf("Cuda malloc Error: %s\n", cudaGetErrorString(cudaStat));
        
        
        for(int j = 0; j < m; j++)
        {
            gettimeofday(&copyTime, NULL);
            avgCopTemp = (((double) copyTime.tv_sec) + ((double) copyTime.tv_usec)/1000000);
            //Fill tileA & tileB with elements from matrix A
            //             printf("Tile A iteration: i=%d, j=%d\n", i,j);
            fillA(Ta, a, tileLength, i, j, m);
            //             printf("Tile B iteration: i=%d, j=%d\n", i,j);
            fillB(Tb, b, tileLength, i, j, m);
            //memcpy TileA and TileB froim host to device
            cudaStat = hipMemcpy(d_a,Ta,tileLength*tileLength*sizeof(float),hipMemcpyHostToDevice);
            if(cudaStat != hipSuccess)
                printf("Cuda memcpy: %s\n", hipGetErrorString(cudaStat));
            cudaStat = hipMemcpy(d_b, Tb, tileLength*tileLength*sizeof(float),hipMemcpyHostToDevice);
            if(cudaStat != hipSuccess)
                printf("Cuda memcpy Error: %s\n", hipGetErrorString(cudaStat));
            //             stat = cublasSetMatrix(tileLength,tileLength,sizeof(*Ta),Ta,tileLength,d_a,tileLength);
            //             if(stat != CUBLAS_STATUS_SUCCESS)
            //                 printf("Cublas to Matrix A Error: %s\n", cublasGetErrorString(stat));
            //             stat = cublasSetMatrix(tileLength,tileLength,sizeof(*Tb),Tb,tileLength,d_b,tileLength);
            //             if(stat != CUBLAS_STATUS_SUCCESS)
            //                 printf("Cublas to Matrix B Error: %s\n", cublasGetErrorString(stat));
            //             stat = cublasSetMatrix(tileLength,tileLength,sizeof(*Tc),Tc,tileLength,d_c,tileLength);
            //             if(stat != CUBLAS_STATUS_SUCCESS)
            //                 printf("Cublas to Matrix C Error: %s\n", cublasGetErrorString(stat));
            gettimeofday(&copyTime, NULL);
            copy_elapsed_time += (((double) copyTime.tv_sec) + ((double) copyTime.tv_usec)/1000000) - avgCopTemp;
//             copy_elapsed_time = copy_elapsed_time + avgCopTemp;
            
            gettimeofday(&mathTime, NULL);
            avgMathTemp = (((double) mathTime.tv_sec) + ((double) mathTime.tv_usec)/1000000);
            stat = hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,tileLength,tileLength,tileLength,&al,d_a,tileLength,d_b,tileLength,&bet,d_c,tileLength);
            if(stat != HIPBLAS_STATUS_SUCCESS)
                printf("Cublas Matrix Multiplication Error: %s\n", cublasGetErrorString(stat));
            //             stat = cublasGetMatrix (tileLength,tileLength, sizeof(*Tc),d_c,tileLength,c,tileLength); // cp d_c - >c
            gettimeofday(&mathTime, NULL);
            math_elapsed_time += (((double) mathTime.tv_sec) + ((double) mathTime.tv_usec)/1000000) - avgMathTemp;
            
            
            hipMemcpy(Tc,d_c, tileLength*tileLength*sizeof(float),hipMemcpyDeviceToHost);
            storeC(Tc,c, tileLength, i, j, m);
            ++numberofIter;
        }
    }
    printf("Copy Execution time : %lf sec.\n", (copy_elapsed_time/numberofIter));
    printf("Math Execution time : %lf sec.\n", math_elapsed_time/numberofIter);
//     printf("numiter %d:\n", timingNumIterations);
//     printf("Copy Execution time : %lf sec.\n", copy_elapsed_time/(timingNumIterations));
    //Free device and host memory for next iteration
    cudaStat = hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);
    free(Ta);
    free(Tb);
    free(Tc);
}







/**********************************************************************
 * function name: main
 * description: test and compare8
 * parameters: 
 * none
 * return: none
 **********************************************************************/
int main(int argc, char** argv) {
    //     cublasStatus_t stat; // CUBLAS functions statusx
    //     cublasHandle_t handle; // CUBLAS context
    int m=20000;// a - mxk matrix
    int n=20000;// b - kxn matrix
    int k=20000;// c - mxn matrix
    // Set status variables
    struct timeval totalTime;
    double total_elapsed_time;
    struct timeval time;
    double elapsed_time;
    
    gettimeofday(&totalTime, NULL);
    total_elapsed_time = (((double) totalTime.tv_sec) + ((double) totalTime.tv_usec)/1000000);
    // Allocate memory in host RAM
    float *a; // mxk matrix a on the host
    float *b; // kxn matrix b on the host
    float *c; // mxn matrix c on the host
    a = (float*) malloc(m*k* sizeof(float)); // host memory for a
    b = (float*) malloc(k*n* sizeof(float)); // host memory for b
    c = (float*) malloc(m*n* sizeof(float)); // host memory for c
    
    /* Assign Random Variables to the matrecies */
    //     srand(3333);
    int val = 1;
    // random initialize matrix A [mxk]
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] =val++;
        }
    }
    val = 1;
    // random initialize matrix B [kxn]
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            b[i * k + j] = val++;
        }
    }
    gettimeofday(&time, NULL);
    elapsed_time = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000);
    
    matrixCpy(a,b,c,10000,2);
    
    gettimeofday(&time, NULL);
    elapsed_time = (((double) time.tv_sec) + ((double) time.tv_usec)/1000000) - elapsed_time;
    printf("Computation Execution time : %lf sec.\n", elapsed_time);
    
    //     int i,j;
    // print matrix A
    //     printf("matA matrix: \n");
    //     for (i = 0; i < m; i++) {
    //         for (j = 0; j < n; j++) {
    //             //printf("[%d][%d]:%d, ", i, j, a[i*k + j]);
    //             printf(" %f ", a[i*k + j]);
    //         }
    //         printf("\n");
    //     }
    //     // print matrix B
    //     printf("\nmatB matrix: \n");
    //     for (i = 0; i < n; i++) {
    //         for (j = 0; j < k; j++) {
    //             //printf("[%d][%d]:%d, ", i, j, b[i*k + j]);
    //             printf(" %f ", b[i*k + j]);
    //         }
    //         printf("\n");
    //     }
    //     // print result matrix
    //     printf("\nResult matrix: \n");
    //     for (i = 0; i < m; i++) {
    //         for (j = 0; j < k; j++) {
    //             //printf("[%d][%d]:%d, ", i, j, c[i*k + j]);
    //             printf(" %f ", c[i*k + j]);
    //         }
    //         printf("\n");
    //     }
    // free memory
    free(a);
    free(b);
    free(c);
    gettimeofday(&totalTime, NULL);
    total_elapsed_time = (((double) totalTime.tv_sec) + ((double) totalTime.tv_usec)/1000000) - total_elapsed_time;
    printf("Execution Total Time : %lf sec.\n", total_elapsed_time);
        
    return 0;
}
