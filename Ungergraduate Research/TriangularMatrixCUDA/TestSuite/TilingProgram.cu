/*
 *  file name: TilingMatrix.cu
 *  NOTE: 
 *       squareMatrixMult is much more efficent than the regular multiplier
 *       currently compiling with: nvcc TilingMatrix.cu -o tileTest
 *       Device Standards for: GeForce GTX 1060 6GB
 *          total global mem size: 6078 MBytes (6373572608 bytes)
 *          total shared mem per block: 49.152 KBytes (49152 bytes)
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include <stdlib.h>

#include <sys/time.h>     //measuring performance data

#define BLOCK_SIZE 32

/**********************************************************************
function name: matrixTriUpper
description: sets a matrix to an upper bound triangle matrix
parameters: 
            &a GPU device pointer to a m X n matrix (A)
Note:
    
return: none
**********************************************************************/
__global__ void matrixTriUpper(float *a, int m, int n) {
    //setting matricies to their upper bound 
    for(int i = 0; i < m; ++i) {
        for(int j = 0; j < n; ++j) {
            if(i>j)
                a[i*n + j] = 0;
            a[i*n + j] = a[i*n + j];
        }
    }
}

/**********************************************************************
function name: matrixMult
description: dot product of two matrix (not only square)
parameters: 
            &a GPU device pointer to a m X n matrix (A)
            &b GPU device pointer to a n X k matrix (B)
            &c GPU device output purpose pointer to a m X k matrix (C) 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
**********************************************************************/
__global__ void matrixMult(float *a, float *b, float *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float  sum = 0;
    if( col < k && row < m) {
        for(int i = 0; i < n; i++) 
            sum += a[row * n + i] * b[i * k + col];
        c[row * k + col] = sum;
    }
} 

/**********************************************************************
function name: squareMatrixMult
description: dot product of two matrix (not only square) in GPU
parameters: 
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C) 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
    SQUARE IS MUCH MORE EFFICENT THAN REGULAR
return: none
**********************************************************************/
__global__ void squareMatrixMult(float *d_a, float *d_b, float *d_result, int n) 
{
    __shared__ float tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n) {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n) {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = threadIdx.x/n; k < BLOCK_SIZE; ++k)  {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n) {
        d_result[row * n + col] = tmp;
    }
}

/**********************************************************************
function name: main
description: test and compare
parameters: 
            none
return: none
**********************************************************************/

int main(int argc, char** argv) {
    int printAllMat = 1; // debug flag for printing all of the maticies
    // Set sizes of the matrixes
    int m=15;
    int n=15;
    int k=15;
    
    /* Fixed seed for illustration */
    srand(3333);

    // Allocate memory in host RAM
    float *copyA, *copyB, *copyC;
    hipHostMalloc((void **) &copyA, sizeof(float)*m*n, hipHostMallocDefault); // copied matrix is m x n
    hipHostMalloc((void **) &copyB, sizeof(float)*n*k, hipHostMallocDefault); // copied matrix is n x k
    hipHostMalloc((void **) &copyC, sizeof(float)*m*k, hipHostMallocDefault); // copied matrix is m x k
    
    // float x = (float)rand()/(float)(RAND_MAX/a);
    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            copyA[i * n + j] =((float)rand()/(float)(RAND_MAX)) * 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            copyB[i * k + j] = ((float)rand()/(float)(RAND_MAX)) * 1024;
        }
    }
    
    // Allocate memory space on the device 
    float *matA, *matB, *matC;
    hipMalloc((void **) &matA, sizeof(float)*m*n); // matrix is m x n
    hipMalloc((void **) &matB, sizeof(float)*n*k); // matrix is n x k
    hipMalloc((void **) &matC, sizeof(float)*m*k); // matrix is m x k
    
    // copy matrix A and B from host to device memory
    hipMemcpy(matA, copyA, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(matB, copyB, sizeof(float)*n*k, hipMemcpyHostToDevice);
    
    printf("size of matA %dX%d: %zu bytes\n", m,n,(sizeof(float)*m*n));
    printf("size of matB %dX%d: %zu bytes\n", n,k,(sizeof(float)*n*k));
    printf("size of matC %dX%d: %zu bytes\n", m,k,(sizeof(float)*m*k));
    printf("total bytes allocated to mem: %zu bytes ", ((sizeof(float)*m*n) + (sizeof(float)*n*k)+ (sizeof(float)*m*k)));
    printf("(~%zu MBytes)\n\n", (((sizeof(float)*m*n) + (sizeof(float)*n*k)+ (sizeof(float)*m*k)) / 1000000)); // get megabytes of the allocated arrays

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    printf("Calculating...\n\n");
    // Launch kernel, check if it is a square
    if(m == n && n == k) {
        matrixTriUpper<<<dimGrid, dimBlock>>>(matA, m, n);
        matrixTriUpper<<<dimGrid, dimBlock>>>(matB, n, k);
        squareMatrixMult<<<dimGrid, dimBlock>>>(matA, matB, matC, n); // square, thus only need 1 param to define size
    }
    else { // not a square, thus it needs param to define all sizes
        matrixMult<<<dimGrid, dimBlock>>>(matA, matB, matC, m, n, k);
    }
    
    // Transefr results from device to host 
    hipMemcpy(copyC, matC, sizeof(float)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); //possibly
    //cudaThreadSynchronize();
    
    //prints the matricies
    // printf("[%d][%d]:%d, ", i, j, copyC[i*k + j]); //Another possible way to print the matrix
    //if the debug flag is on it will print the first two product arrays as well
    int i,j;
    if(printAllMat == 1) {
        // print matrix A
        printf("matA matrix: \n");
        for (i = 0; i < m; i++) {
            for (j = 0; j < n; j++) {
                //printf("[%d][%d]:%d, ", i, j, copyA[i*k + j]);
                printf(" %f ", copyA[i*k + j]);
            }
            printf("\n");
        }
        // print matrix B
        printf("\nmatB matrix: \n");
        for (i = 0; i < n; i++) {
            for (j = 0; j < k; j++) {
                //printf("[%d][%d]:%d, ", i, j, copyB[i*k + j]);
                printf(" %f ", copyB[i*k + j]);
            }
            printf("\n");
        }
    }
    
    // print result matrix
    printf("\nResult matrix: \n");
    for (i = 0; i < m; i++) {
        for (j = 0; j < k; j++) {
                //printf("[%d][%d]:%d, ", i, j, copyC[i*k + j]);
                printf(" %f ", copyC[i*k + j]);
            }
        printf("\n");
    }
    
    // free memory
    hipFree(matA);
    hipFree(matB);
    hipFree(matC);
    hipHostFree(copyA);
    hipHostFree(copyB);
    hipHostFree(copyC);
    return 0;
}
